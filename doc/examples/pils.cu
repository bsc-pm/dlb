#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <nvToolsExt.h>

#define DEFAULT_CPU_LOAD 1000000
#define DEFAULT_ACCEL_LOAD 1000000
#define DEFAULT_MEMCPY_SIZE (10 * 1024 * 1024)  // 10 MB

__device__ float gpu_compute_op(float p, int x) {
    for (int i = 1; i < x; i++) {
        p += 0.99999f * i;
        p = p / i;
    }
    return p;
}

__global__ void gpu_computation(float *p, int x, int *iters, long long target_time) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        long long start = clock64();
        long long now;
        int count = 0;
        float val = *p;
        do {
            val = gpu_compute_op(val, x);
            count++;
            now = clock64();
        } while ((now - start) < target_time);
        *p = val;
        *iters = count;
    }
}

long long int cpu_fibonacci(int n) {
    if (n <= 1) return n;
    return cpu_fibonacci(n - 1) + cpu_fibonacci(n - 2);
}

long long usecs() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1000000LL + t.tv_usec;
}

void print_usage(char *prog_name) {
     printf("Usage: %s -n <fib_n> -a <accel_usec,...> -c <cpu_usec,...> -m <mem_usec,...> [-s <buffer_size>]\n", prog_name);
}


void parse_loads(char *str, int *loads, int mpi_size, int default_value) {
    for (int i = 0; i < mpi_size; ++i) {
        loads[i] = default_value;
    }
    if (!str) return;
    char *token = strtok(str, ",");
    for (int i = 0; token && i < mpi_size; ++i) {
        loads[i] = atoi(token);
        token = strtok(NULL, ",");
    }
}


static int cpu_computation(int num, int usec) {
    char range_name[64];
    snprintf(range_name, sizeof(range_name), "Rank CPU Loop");
    nvtxRangeId_t range_id = nvtxRangeStartA(range_name);

    float a = 0.99999f;
    float p = num;
    int x = 145;
    int i;
    long long t_start = usecs();
    long long t_now;
    do {
        for (i = 1; i < x; i++) {
            p += a * i;
            p = p / i;
        }
        t_now = usecs();
    } while ((t_now - t_start) < usec);

    nvtxRangeEnd(range_id);
    return (int)p;
}

static int gpu_timed_computation(int num, int usec) {
    float *d_p;
    int *d_total_iters;
    float h_p = num;
    int h_total_iters = 0;
    int threadsPerBlock = 256;
    int blocksPerGrid = 1;
    long long target_time = usec * 1000LL;

    hipMalloc((void**)&d_p, sizeof(float));
    hipMalloc((void**)&d_total_iters, sizeof(int));
    hipMemcpy(d_p, &h_p, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_total_iters, &h_total_iters, sizeof(int), hipMemcpyHostToDevice);

    printf("DEBUG: Launching kernel with %d blocks, %d threads\n",
           blocksPerGrid, threadsPerBlock);
    fflush(stdout);

    gpu_computation<<<blocksPerGrid, threadsPerBlock>>>(d_p, num, d_total_iters, target_time);
    hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel execution failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }


    hipMemcpy(&h_p, d_p, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_total_iters, d_total_iters, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_p);
    hipFree(d_total_iters);

    return (int)h_p;
}
static void gpu_memcpy_timed(int usec, size_t buffer_size) {
    char *h_buf = (char *)malloc(buffer_size);
    char *d_buf;

    hipMalloc((void**)&d_buf, buffer_size);

    // Initialize host buffer to some data
    memset(h_buf, 1, buffer_size);

    long long t_start = usecs();
    long long t_now;
    do {
        hipMemcpy(d_buf, h_buf, buffer_size, hipMemcpyHostToDevice);
        hipMemcpy(h_buf, d_buf, buffer_size, hipMemcpyDeviceToHost);
        t_now = usecs();
    } while ((t_now - t_start) < usec);

    hipFree(d_buf);
    free(h_buf);
}

int main(int argc, char *argv[]) {
    int provided;
    MPI_Init_thread(&argc, &argv, MPI_THREAD_FUNNELED, &provided);

    int mpi_rank, mpi_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

    int fib_n = 35;
    char *accel_str = NULL;
    char *cpu_str = NULL;
    char *mem_str = NULL;
    size_t buffer_size = DEFAULT_MEMCPY_SIZE;
    double mem_time = 0.0;

    int opt;
        while ((opt = getopt(argc, argv, "n:a:c:m:s:")) != -1) {
        switch (opt) {
            case 'n': fib_n = atoi(optarg); break;
            case 'a': accel_str = strdup(optarg); break;
            case 'c': cpu_str = strdup(optarg); break;
            case 'm': mem_str = strdup(optarg); break;
            case 's': buffer_size = atol(optarg); break;
            default:
                if (mpi_rank == 0) print_usage(argv[0]);
                MPI_Finalize();
                return 1;
        }
    }
   

    int *cpu_loads = (int *)calloc(mpi_size, sizeof(int));
    int *accel_loads = (int *)calloc(mpi_size, sizeof(int));
    int *mem_loads = (int *)calloc(mpi_size, sizeof(int));


    parse_loads(cpu_str, cpu_loads, mpi_size, DEFAULT_CPU_LOAD);
    parse_loads(accel_str, accel_loads, mpi_size, DEFAULT_ACCEL_LOAD);
    parse_loads(mem_str, mem_loads, mpi_size, 0);  // Default to 0 (no memory ops)

    double cpu_time = 0.0, accel_time = 0.0;
    long long t_start, t_end;

    if (accel_loads[mpi_rank] > 0) {
        char range_name[64];
        snprintf(range_name, sizeof(range_name), "Rank %d GPU Load", mpi_rank);
        nvtxRangeId_t id = nvtxRangeStartA(range_name);

        t_start = usecs();
        gpu_timed_computation(fib_n, accel_loads[mpi_rank]);
        t_end = usecs();

        nvtxRangeEnd(id);
        accel_time = (t_end - t_start) / 1e6;
	hipDeviceSynchronize(); // Ensure kernels complete
    }

    if (cpu_loads[mpi_rank] > 0) {
        char range_name[64];
        snprintf(range_name, sizeof(range_name), "Rank %d CPU Load", mpi_rank);
        nvtxRangeId_t id = nvtxRangeStartA(range_name);

        t_start = usecs();
        cpu_computation(fib_n, cpu_loads[mpi_rank]);
        t_end = usecs();

        nvtxRangeEnd(id);
        cpu_time = (t_end - t_start) / 1e6;
    }
    if (mem_loads[mpi_rank] > 0) {
        char range_name[64];
        snprintf(range_name, sizeof(range_name), "Rank %d Memory Copy", mpi_rank);
        nvtxRangeId_t id = nvtxRangeStartA(range_name);

        t_start = usecs();
        gpu_memcpy_timed(mem_loads[mpi_rank], buffer_size);
        t_end = usecs();

        nvtxRangeEnd(id);
        mem_time = (t_end - t_start) / 1e6;
    }
    printf("Rank %d completed CPU dummy computation in %.3f s, Accelerator dummy computation in %.3f s, and Memory copies in %.3f s\n",
           mpi_rank, cpu_time, accel_time, mem_time);

    // Synchronize all ranks before finalizing
    MPI_Barrier(MPI_COMM_WORLD);

    free(cpu_loads);
    free(accel_loads);
    free(mem_loads);
    if (mem_str) free(mem_str);
    if (cpu_str) free(cpu_str);
    if (accel_str) free(accel_str);

    MPI_Finalize();
    return 0;
}
